#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <fstream>		// ifstream
#include <ctime>		// time(NULL)
#include <random>		// C++11 Mersenne twister
#include <climits>		// UINT_MAX
#include <cmath>		// log, cos, sin, ceil, M_PI
#include <algorithm>	// min
#include <cstdio>

#include "libraries/InputStructures/InputGPUData/Input_gpu_data.cuh"
#include "libraries/InputStructures/InputMarketData/Input_market_data.cuh"
#include "libraries/InputStructures/InputMCData/Input_MC_data.cuh"
#include "libraries/InputStructures/InputOptionData/Input_option_data.cuh"
#include "libraries/CoreLibraries/DataStreamManager/Data_stream_manager.cuh"
#include "libraries/CoreLibraries/Statistics/Statistics.cuh"
#include "libraries/CoreLibraries/Path/Path.cuh"
#include "libraries/CoreLibraries/RandomGenerator/rng.cuh"
#include "libraries/CoreLibraries/SupportFunctions/Support_functions.cuh"
#include "libraries/OutputStructures/OutputMCData/Output_MC_data.cuh"

using namespace std;

int main(){
	
	// Read & print input data from file
	Data_stream_manager streamManager("input.dat");
	
	Input_gpu_data inputGPU;
	Input_market_data inputMarket;
	Input_option_data inputOption;
	Input_MC_data inputMC;
	streamManager.ReadInputData(inputGPU, inputOption, inputMarket, inputMC);
	
	unsigned int numberOfThreadsPerBlock = inputGPU.GetNumberOfThreadsPerBlock();
	unsigned int totalNumberOfThreads = inputGPU.GetTotalNumberOfThreads();
	unsigned int numberOfSimulationsPerThread = inputMC.GetNumberOfSimulationsPerThread(inputGPU);
	
	streamManager.PrintInputData(inputGPU, inputOption, inputMarket, inputMC);
															
	// Output arrays
	Statistics *exactOutputs = new Statistics[totalNumberOfThreads];
	Statistics *eulerOutputs = new Statistics[totalNumberOfThreads];

/*
	////////////// HOST-SIDE GENERATOR //////////////	
	cout << "Beginning device simulation through CPU..." << endl;
	// Simulating device function
	OptionPricingEvaluator_Host(inputGPU, inputOption, inputMarket, inputMC, exactOutputs, eulerOutputs);
	cout << endl;
	/////////////////////////////////////////////////
*/

///*
	////////////// DEVICE-SIDE GENERATOR //////////////
	Statistics *device_exactOutputs;
	Statistics *device_eulerOutputs;
	
	hipMalloc((void **)&device_exactOutputs, totalNumberOfThreads*sizeof(Statistics));
	hipMalloc((void **)&device_eulerOutputs, totalNumberOfThreads*sizeof(Statistics));
	
	hipMemcpy(device_exactOutputs, exactOutputs, totalNumberOfThreads*sizeof(Statistics), hipMemcpyHostToDevice);
	hipMemcpy(device_eulerOutputs, eulerOutputs, totalNumberOfThreads*sizeof(Statistics), hipMemcpyHostToDevice);

	cout << "Beginning GPU computation..." << endl;
	OptionPricingEvaluator_Global<<<inputGPU.NumberOfBlocks,numberOfThreadsPerBlock>>>(inputGPU, inputOption, inputMarket, inputMC, device_exactOutputs, device_eulerOutputs);

	hipMemcpy(exactOutputs, device_exactOutputs, totalNumberOfThreads*sizeof(Statistics), hipMemcpyDeviceToHost);
	hipMemcpy(eulerOutputs, device_eulerOutputs, totalNumberOfThreads*sizeof(Statistics), hipMemcpyDeviceToHost);

	hipFree(device_exactOutputs);
	hipFree(device_eulerOutputs);
	///////////////////////////////////////////////////
//*/
	
	// Compute results
	Statistics exactResults;
	Statistics eulerResults;
	
	for(unsigned int threadNumber=0; threadNumber<totalNumberOfThreads; ++threadNumber){
		exactResults += exactOutputs[threadNumber];
		eulerResults += eulerOutputs[threadNumber];
	}
	
	exactResults.EvaluateEstimatedPriceAndError();
	eulerResults.EvaluateEstimatedPriceAndError();
	
	// Elapsed time is temporary, will be implemented later
	double elapsedTime = 0.;
	
	// Global output MC
	Output_MC_data outputMC;
	streamManager.StoreOutputData(outputMC, exactResults, eulerResults, elapsedTime);
	streamManager.PrintOutputData(outputMC);
	
	// Trash bin section, where segfaults come to die
	delete[] exactOutputs;
	delete[] eulerOutputs;

	return 0;
}
