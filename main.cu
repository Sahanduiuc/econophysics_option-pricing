#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <fstream>		// ifstream
#include <ctime>		// time(NULL)
#include <random>		// C++11 Mersenne twister
#include <climits>		// UINT_MAX
#include <cmath>		// log, cos, sin, ceil, M_PI
#include <algorithm>	// min
#include <cstdio>

#include "libraries/InputStructures/InputGPUData/Input_gpu_data.cuh"
#include "libraries/InputStructures/InputMarketData/Input_market_data.cuh"
#include "libraries/InputStructures/InputMCData/Input_MC_data.cuh"
#include "libraries/InputStructures/InputOptionData/Input_option_data.cuh"
#include "libraries/CoreLibraries/DataStreamManager/Data_stream_manager.cuh"
#include "libraries/CoreLibraries/Statistics/Statistics.cuh"
#include "libraries/CoreLibraries/Path/Path.cuh"
#include "libraries/CoreLibraries/RandomGenerator/RNG.cuh"
#include "libraries/CoreLibraries/SupportFunctions/Support_functions.cuh"
#include "libraries/OutputStructures/OutputMCData/Output_MC_data.cuh"

using namespace std;

int main(){
	
	hipEvent_t eventStart, eventStop;
	float elapsedTime;
	hipEventCreate(&eventStart);
	hipEventCreate(&eventStop);
	
	// Read & print input data from file
	Data_stream_manager streamManager("input.dat");
	
	Input_gpu_data inputGPU;
	Input_market_data inputMarket;
	Input_option_data inputOption;
	Input_MC_data inputMC;
	streamManager.ReadInputData(inputGPU, inputOption, inputMarket, inputMC);
	
	unsigned int numberOfThreadsPerBlock = inputGPU.GetNumberOfThreadsPerBlock();
	unsigned int totalNumberOfThreads = inputGPU.GetTotalNumberOfThreads();
	unsigned int numberOfSimulationsPerThread = inputMC.GetNumberOfSimulationsPerThread(inputGPU);
	
	streamManager.PrintInputData(inputGPU, inputOption, inputMarket, inputMC);
															
	// Output arrays
	Statistics *exactOutputs = new Statistics[totalNumberOfThreads];
	Statistics *eulerOutputs = new Statistics[totalNumberOfThreads];
	
	// Seed for random number generation
	// Fix it to a value between 129 and UINT_MAX-totalNumberOfThreads or let time(NULL) do its magic
	unsigned int seed;	
	do
		seed = time(NULL);
	while(seed < 129 || seed > UINT_MAX - totalNumberOfThreads);


	hipEventRecord(eventStart,0);

///*
	////////////// HOST-SIDE GENERATOR //////////////	
	cout << "Beginning device simulation through CPU..." << endl;
	// Simulating device function
	OptionPricingEvaluator_Host(inputGPU, inputOption, inputMarket, inputMC, exactOutputs, eulerOutputs, seed);
	cout << endl;
	/////////////////////////////////////////////////
//*/

/*
	////////////// DEVICE-SIDE GENERATOR //////////////
	Statistics *device_exactOutputs;
	Statistics *device_eulerOutputs;
	
	hipMalloc((void **)&device_exactOutputs, totalNumberOfThreads*sizeof(Statistics));
	hipMalloc((void **)&device_eulerOutputs, totalNumberOfThreads*sizeof(Statistics));
	
	hipMemcpy(device_exactOutputs, exactOutputs, totalNumberOfThreads*sizeof(Statistics), hipMemcpyHostToDevice);
	hipMemcpy(device_eulerOutputs, eulerOutputs, totalNumberOfThreads*sizeof(Statistics), hipMemcpyHostToDevice);

	cout << "Beginning GPU computation..." << endl;
	OptionPricingEvaluator_Global<<<inputGPU.NumberOfBlocks,numberOfThreadsPerBlock>>>(inputGPU, inputOption, inputMarket, inputMC, device_exactOutputs, device_eulerOutputs, seed);

	hipMemcpy(exactOutputs, device_exactOutputs, totalNumberOfThreads*sizeof(Statistics), hipMemcpyDeviceToHost);
	hipMemcpy(eulerOutputs, device_eulerOutputs, totalNumberOfThreads*sizeof(Statistics), hipMemcpyDeviceToHost);

	hipFree(device_exactOutputs);
	hipFree(device_eulerOutputs);
	///////////////////////////////////////////////////
*/

	hipEventRecord(eventStop,0);
	hipEventSynchronize(eventStop);
	hipEventElapsedTime(&elapsedTime, eventStart, eventStop);
	
	// Compute results
	Statistics exactResults;
	Statistics eulerResults;
	
	for(unsigned int threadNumber=0; threadNumber<totalNumberOfThreads; ++threadNumber){
		exactResults += exactOutputs[threadNumber];
		eulerResults += eulerOutputs[threadNumber];
	}
	
	exactResults.EvaluateEstimatedPriceAndError();
	eulerResults.EvaluateEstimatedPriceAndError();
	
	// Global output MC
	Output_MC_data outputMC;
	streamManager.StoreOutputData(outputMC, exactResults, eulerResults, elapsedTime);
	streamManager.PrintOutputData(outputMC);
	
	// Trash bin section, where segfaults come to die
	delete[] exactOutputs;
	delete[] eulerOutputs;
	
	hipEventDestroy(eventStart);
	hipEventDestroy(eventStop);

	return 0;
}
