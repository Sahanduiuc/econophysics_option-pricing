#include "hip/hip_runtime.h"
#include <cmath>	// fabs, sqrt, pow

#include "Output_MC_data.cuh"

__device__ __host__ double Output_MC_data::GetRelativeErrorEuler() const{
	return this->ErrorMCEuler / this->EstimatedPriceMCEuler;
}

__device__ __host__ double Output_MC_data::GetRelativeErrorExact() const{
	return this->ErrorMCExact / this->EstimatedPriceMCExact;
}

__device__ __host__ double Output_MC_data::GetEulerToExactDiscrepancy() const{
	return fabs((this->EstimatedPriceMCEuler - this->EstimatedPriceMCExact)/sqrt(pow(this->ErrorMCEuler,2)+pow(ErrorMCExact,2)));
}
