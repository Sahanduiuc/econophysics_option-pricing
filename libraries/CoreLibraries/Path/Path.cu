#include "hip/hip_runtime.h"
#include <cmath>	// sqrt, pow, fmax, log, exp, fabs

#include "Path.cuh"
#include "../../InputStructures/InputMarketData/Input_market_data.cuh"
#include "../../InputStructures/InputOptionData/Input_option_data.cuh"

using namespace std;

// Constructors
__device__ __host__ Path::Path(){
	this->_OptionType = NULL;
	this->_SpotPrice = 0.;
	this->_RiskFreeRate = NULL;
	this->_Volatility = NULL;
	this->_InitialPrice = NULL;
	this->_TimeToMaturity = NULL;
	this->_NumberOfIntervals = NULL;
	this->_DeltaTime = 0.;
	this->_StrikePrice = NULL;
	this->_B = NULL;
	this->_N = NULL;
	this->_K = NULL;
	this->_PerformanceCorridorBarrierCounter = 0;
	this->_NegativePrice = false;
}

__device__ __host__ Path::Path(const Input_market_data& market, const Input_option_data& option){
	this->_OptionType = &(option.OptionType);
	this->_SpotPrice = market.InitialPrice;
	this->_RiskFreeRate = &(market.RiskFreeRate);
	this->_Volatility = &(market.Volatility);
	this->_InitialPrice = &(market.InitialPrice);
	this->_TimeToMaturity = &(option.TimeToMaturity);
	this->_NumberOfIntervals = &(option.NumberOfIntervals);
	this->_DeltaTime = option.GetDeltaTime();
	this->_StrikePrice = &(option.StrikePrice);
	this->_B = &(option.B);
	this->_N = &(option.N);
	this->_K = &(option.K);
	this->_PerformanceCorridorBarrierCounter = 0;
	this->_NegativePrice = false;
}

// Public set methods
__device__ __host__ void Path::ResetToInitialState(const Input_market_data& market, const Input_option_data& option){
	this->_OptionType = &(option.OptionType);
	this->_SpotPrice = market.InitialPrice;
	this->_RiskFreeRate = &(market.RiskFreeRate);
	this->_Volatility = &(market.Volatility);
	this->_InitialPrice = &(market.InitialPrice);
	this->_TimeToMaturity = &(option.TimeToMaturity);
	this->_NumberOfIntervals = &(option.NumberOfIntervals);
	this->_DeltaTime = option.GetDeltaTime();
	this->_StrikePrice = &(option.StrikePrice);
	this->_B = &(option.B);
	this->_N = &(option.N);
	this->_K = &(option.K);
	this->_PerformanceCorridorBarrierCounter = 0;
	this->_NegativePrice = false;
}

__device__ __host__ void Path::ResetToInitialState(const Path& otherPath){
	this->_OptionType = otherPath._OptionType;
	this->_SpotPrice = otherPath._SpotPrice;
	this->_RiskFreeRate = otherPath._RiskFreeRate;
	this->_Volatility = otherPath._Volatility;
	this->_InitialPrice = otherPath._InitialPrice;
	this->_TimeToMaturity = otherPath._TimeToMaturity;
	this->_NumberOfIntervals = otherPath._NumberOfIntervals;
	this->_DeltaTime = otherPath._DeltaTime;
	this->_StrikePrice = otherPath._StrikePrice;
	this->_B = otherPath._B;
	this->_N = otherPath._N;
	this->_K = otherPath._K;
	this->_PerformanceCorridorBarrierCounter = otherPath._PerformanceCorridorBarrierCounter;
	this->_NegativePrice = otherPath._NegativePrice;
}

// Public get methods
__device__ __host__ double Path::GetSpotPrice() const{
	return this->_SpotPrice;
}

__device__ __host__ unsigned int Path::GetPerformanceCorridorBarrierCounter() const{
	return this->_PerformanceCorridorBarrierCounter;
}


// Euler and exact steps implementation
__device__ __host__ void Path::EulerLogNormalStep(double gaussianRandomVariable){
	double SpotPrice_i;		//The price at the next step
	SpotPrice_i = (this->_SpotPrice) *
	(1 + *(this->_RiskFreeRate) * this->_DeltaTime
	+ *(this->_Volatility) * sqrt(this->_DeltaTime) * gaussianRandomVariable);
	
	if(*(_OptionType) == 'e')
		this->CheckPerformanceCorridorCondition(this->_SpotPrice, SpotPrice_i);
		
	if(SpotPrice_i < 0)
		this->_NegativePrice = true;
	
	this->_SpotPrice = SpotPrice_i;
}

__device__ __host__ void Path::ExactLogNormalStep(double gaussianRandomVariable){
	double SpotPrice_i;		//The price at the next step
	SpotPrice_i = (this->_SpotPrice) * exp((*(this->_RiskFreeRate)
	- 0.5 * pow(*(this->_Volatility),2)) * this->_DeltaTime
	+ *(this->_Volatility) * gaussianRandomVariable * sqrt(this->_DeltaTime));
	
	if(*(_OptionType) == 'e')
		this->CheckPerformanceCorridorCondition(this->_SpotPrice, SpotPrice_i);
	
	this->_SpotPrice = SpotPrice_i;
}

// Check performance corridor condition
__device__ __host__ void Path::CheckPerformanceCorridorCondition(double currentSpotPrice, double nextSpotPrice){
	double modulusArgument = 1./(sqrt(this->_DeltaTime)) * log(nextSpotPrice / currentSpotPrice);
	double barrier = *(this->_B) * *(this->_Volatility);

	if(fabs(modulusArgument) < barrier)
		++(this->_PerformanceCorridorBarrierCounter);
}

// Evaluate atualized payoff
__device__ __host__ double Path::GetActualizedPayoff() const{
	double payoff;
	
	switch(*(this->_OptionType)){
		case 'f':
			payoff = this->_SpotPrice;
			break;
		
		case 'c':
			payoff = fmax(this->_SpotPrice - *(this->_StrikePrice), 0.);
			break;
		
		case 'p':
			payoff = fmax(*(this->_StrikePrice) - this->_SpotPrice, 0.);
			break;
		
		case 'e':
			payoff = *(this->_N) * fmax((static_cast<double>(this->_PerformanceCorridorBarrierCounter) / *(this->_NumberOfIntervals)) - *(this->_K), 0.);
			break;
			
		default:
			payoff = -10000.;
			break;
	}	
	
	return (payoff * exp(- *(this->_RiskFreeRate) * *(this->_TimeToMaturity)));
}

__device__ __host__ bool Path::GetNegativePrice() const{
	return this->_NegativePrice;
}

__device__ __host__ double Path::GetBlackAndScholesPrice() const{
	double d1 = 1./(*(this->_Volatility) * sqrt(*(this->_TimeToMaturity))) 
	* (log(*(this->_InitialPrice) / *(this->_StrikePrice))
	+ (*(this->_RiskFreeRate) + pow(*(this->_Volatility),2)/2) * *(this->_TimeToMaturity));

	double d2 = d1 -  *(this->_Volatility) * sqrt(*(this->_TimeToMaturity));

	if(*(this->_OptionType) == char('c')){
		double callPrice = *(this->_InitialPrice) * (0.5 * (1. + erf(d1/sqrt(2.)))) - *(this->_StrikePrice) 
		* exp(- *(this->_RiskFreeRate) * *(this->_TimeToMaturity))
		* (0.5 * (1. + erf(d2/sqrt(2.))));

		return callPrice;
	} 

	if(*(this->_OptionType) == char('p')){
		double putPrice = *(this->_InitialPrice) * ((0.5 * (1. + erf(d1/sqrt(2.)))) - 1) - *(this->_StrikePrice)
		* exp(- *(this->_RiskFreeRate) * *(this->_TimeToMaturity))
		* ((0.5 * (1. + erf(d2/sqrt(2.)))) - 1);

		return putPrice;
	}
}
