#include "hip/hip_runtime.h"
#include "Path.cuh"
#include <cmath>

using namespace std;

// Constructors
__device__ __host__ Path::Path(){
	this->_OptionType = 'c';
	this->_SpotPrice = 0.;
	this->_RiskFreeRate = 0.;
	this->_Volatility = 0.;
	this->_TimeToMaturity = 0;
	this->_NumberOfIntervals = 0;
	this->_DeltaTime = 0.;
	this->_StrikePrice = 0.;
	this->_B = 0.;
	this->_N = 0.;
	this->_K = 0.;
	this->_PerformanceCorridorBarrierCounter = 0;
}

__device__ __host__ Path::Path(const Input_market_data& market, const Input_option_data& option){
	this->_OptionType = option.OptionType;
	this->_SpotPrice = market.InitialPrice;
	this->_RiskFreeRate = market.RiskFreeRate;
	this->_Volatility = market.Volatility;
	this->_TimeToMaturity = option.TimeToMaturity;
	this->_NumberOfIntervals = option.NumberOfIntervals;
	this->_DeltaTime = option.GetDeltaTime();
	this->_StrikePrice = option.StrikePrice;
	this->_B = option.B;
	this->_N = option.N;
	this->_K = option.K;
	this->_PerformanceCorridorBarrierCounter = 0;
}

// Public set methods
__device__ __host__ void Path::ResetToInitialState(const Input_market_data& market, const Input_option_data& option){
	this->_OptionType = option.OptionType;
	this->_SpotPrice = market.InitialPrice;
	this->_RiskFreeRate = market.RiskFreeRate;
	this->_Volatility = market.Volatility;
	this->_TimeToMaturity = option.TimeToMaturity;
	this->_NumberOfIntervals = option.NumberOfIntervals;
	this->_DeltaTime = option.GetDeltaTime();
	this->_StrikePrice = option.StrikePrice;
	this->_B = option.B;
	this->_N = option.N;
	this->_K = option.K;
	this->_PerformanceCorridorBarrierCounter = 0;
}

__device__ __host__ void Path::ResetToInitialState(const Path& otherPath){
	this->_OptionType = otherPath._OptionType;
	this->_SpotPrice = otherPath._SpotPrice;
	this->_RiskFreeRate = otherPath._RiskFreeRate;
	this->_Volatility = otherPath._Volatility;
	this->_TimeToMaturity = otherPath._TimeToMaturity;
	this->_NumberOfIntervals = otherPath._NumberOfIntervals;
	this->_DeltaTime = otherPath._DeltaTime;
	this->_StrikePrice = otherPath._StrikePrice;
	this->_B = otherPath._B;
	this->_N = otherPath._N;
	this->_K = otherPath._K;
	this->_PerformanceCorridorBarrierCounter = otherPath._PerformanceCorridorBarrierCounter;
}

// Public get methods
__device__ __host__ double Path::GetSpotPrice() const{
	return this->_SpotPrice;
}

__device__ __host__ unsigned int Path::GetPerformanceCorridorBarrierCounter() const{
	return this->_PerformanceCorridorBarrierCounter;
}


// Euler and exact steps implementation
__device__ __host__ void Path::EulerLogNormalStep(double gaussianRandomVariable){
	double SpotPrice_i;		//The price at the next step
	SpotPrice_i = (this->_SpotPrice) *
	(1 + this->_RiskFreeRate * this->_DeltaTime
	+ this->_Volatility * sqrt(this->_DeltaTime) * gaussianRandomVariable);
	
	if(_OptionType == 'e')
		this->CheckPerformanceCorridorCondition(this->_SpotPrice, SpotPrice_i);
	
	this->_SpotPrice = SpotPrice_i;
}

__device__ __host__ void Path::ExactLogNormalStep(double gaussianRandomVariable){
	double SpotPrice_i;		//The price at the next step
	SpotPrice_i = (this->_SpotPrice) * exp((this->_RiskFreeRate
	- 0.5 * pow(this->_Volatility,2)) * this->_DeltaTime
	+ this->_Volatility * gaussianRandomVariable * sqrt(this->_DeltaTime));
	
	if(_OptionType == 'e')
		this->CheckPerformanceCorridorCondition(this->_SpotPrice, SpotPrice_i);
	
	this->_SpotPrice = SpotPrice_i;
}

// Check performance corridor condition
__device__ __host__ void Path::CheckPerformanceCorridorCondition(double currentSpotPrice, double nextSpotPrice){
	double modulusArgument = 1./(sqrt(this->_DeltaTime)) * log(nextSpotPrice / currentSpotPrice);
	double barrier = this->_B * this->_Volatility;

	if(fabs(modulusArgument) < barrier)
		++(this->_PerformanceCorridorBarrierCounter);
}

// Evaluate atualized payoff
__device__ __host__ double Path::GetActualizedPayoff() const{
	double payoff;
	
	switch(this->_OptionType){
		case 'f':
			payoff = this->_SpotPrice;
			break;
		
		case 'c':
			payoff = fmax(this->_SpotPrice - this->_StrikePrice, 0.);
			break;
		
		case 'p':
			payoff = fmax(this->_StrikePrice - this->_SpotPrice, 0.);
			break;
		
		case 'e':
			payoff = this->_N * fmax((static_cast<double>(this->_PerformanceCorridorBarrierCounter) / this->_NumberOfIntervals) - this->_K, 0.);
			break;
			
		default:
			payoff = -10000.;
			break;
	}	
	
	return (payoff * exp(- this->_RiskFreeRate * this->_TimeToMaturity));
}
