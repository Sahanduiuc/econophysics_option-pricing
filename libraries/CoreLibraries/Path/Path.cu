#include "hip/hip_runtime.h"
#include "Path.cuh"
#include <cmath>

using namespace std;

// Constructors
__device__ __host__ Path::Path(const Input_market_data& market, const Input_option_data& option, double SpotPrice){
	this->_SpotPrice = SpotPrice;
	this->_RiskFreeRate = market._RiskFreeRate;
	this->_Volatility = market._Volatility;
	this->_DeltaTime = option.GetDeltaTime();
	
	this->_B = 0;
	this->_N = 0;
	this->_K = 0;
	this->_PerformanceCorridorBarrierCounter = 0;
}

__device__ __host__ Path::Path(const Input_market_data& market, const Input_option_data_PlainVanilla& option, double SpotPrice){
	this->_SpotPrice = SpotPrice;
	this->_RiskFreeRate = market._RiskFreeRate;
	this->_Volatility = market._Volatility;
	this->_DeltaTime = option.GetDeltaTime();
	
	this->_B = 0;
	this->_N = 0;
	this->_K = 0;
	this->_PerformanceCorridorBarrierCounter = 0;
}
__device__ __host__ Path::Path(const Input_market_data& market, const Input_option_data_PerformanceCorridor& option, double SpotPrice){
	this->_SpotPrice = SpotPrice;
	this->_RiskFreeRate = market._RiskFreeRate;
	this->_Volatility = market._Volatility;
	this->_DeltaTime = option.GetDeltaTime();
	this->_B = option._B;
	this->_N = option._N;
	this->_K = option._K;
	this->_PerformanceCorridorBarrierCounter = 0;
}

// Public set methods
__device__ __host__ void Path::SetInternalData(const Input_market_data& market, const Input_option_data& option, double SpotPrice){
	this->_SpotPrice = SpotPrice;
	this->_RiskFreeRate = market._RiskFreeRate;
	this->_Volatility = market._Volatility;
	this->_DeltaTime = option.GetDeltaTime();

	this->_B = 0;
	this->_N = 0;
	this->_K = 0;
	this->_PerformanceCorridorBarrierCounter = 0;
}

__device__ __host__ void Path::SetInternalData(const Input_market_data& market, const Input_option_data_PlainVanilla& option, double SpotPrice){
	this->_SpotPrice = SpotPrice;
	this->_RiskFreeRate = market._RiskFreeRate;
	this->_Volatility = market._Volatility;
	this->_DeltaTime = option.GetDeltaTime();

	this->_B = 0;
	this->_N = 0;
	this->_K = 0;
	this->_PerformanceCorridorBarrierCounter = 0;
}

__device__ __host__ void Path::SetInternalData(const Input_market_data& market, const Input_option_data_PerformanceCorridor& option, double SpotPrice){
	this->_SpotPrice = SpotPrice;
	this->_RiskFreeRate = market._RiskFreeRate;
	this->_Volatility = market._Volatility;
	this->_DeltaTime = option.GetDeltaTime();
	this->_B = option._B;
	this->_N = option._N;
	this->_K = option._K;
	this->_PerformanceCorridorBarrierCounter = 0;
}

__device__ __host__ void Path::SetInternalData(const Path& otherPath){
	this->_SpotPrice = otherPath._SpotPrice;
	this->_RiskFreeRate = otherPath._RiskFreeRate;
	this->_Volatility = otherPath._Volatility;
	this->_DeltaTime = otherPath._DeltaTime();
	this->_B = otherPath._B;
	this->_N = otherPath._N;
	this->_K = otherPath._K;
	this->_PerformanceCorridorBarrierCounter = otherPath._PerformanceCorridorBarrierCounter;
}

// Public get methods
__device__ __host__ double Path::GetSpotPrice() const{
	return _SpotPrice;
}


// Euler and exact steps implementation
__device__ __host__ void Path::EulerLogNormalStep(double gaussianRandomVariable){
	double SpotPrice_i;		//The price at the next step
	SpotPrice_i = (this->_SpotPrice) *
	(1 + this->_RiskFreeRate * this->_DeltaTime
	+ this->_Volatility * sqrt(this->_DeltaTime) * gaussianRandomVariable);
	
	this->CheckPerformanceCorridorCondition(this->_SpotPrice, SpotPrice_i);
	this->_SpotPrice = SpotPrice_i;
}

__device__ __host__ void Path::ExactLogNormalStep(double gaussianRandomVariable){
	double SpotPrice_i;		//The price at the next step
	SpotPrice_i = (this->_SpotPrice) * exp((this->_RiskFreeRate
	- 0.5 * pow(this->_Volatility,2)) * this->_DeltaTime
	+ this->_Volatility * gaussianRandomVariable * sqrt(this->_DeltaTime));

	this->CheckPerformanceCorridorCondition(this->_SpotPrice, SpotPrice_i);
	this->_SpotPrice = SpotPrice_i;
}

// Check performance corridor condition
void Path::CheckPerformanceCorridorCondition(double currentSpotPrice, double nextSpotPrice){
	modulusArgument = 1./(sqrt(_DeltaTime)) * log(nextSpotPrice / currentSpotPrice);
	barrier = _B * _Volatility;
	
	if(fabs(modulusArgument) < barrier)
		++_PerformanceCorridorBarrierCounter;
}
